#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

__global__ void increment_kernel(int *g_data, int inc_value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] += inc_value;
}

__host__ bool correct_output(int *data, const int n, const int x) {
    for (int i = 0; i < n; ++i) {
        if (data[i] != x) {
            printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
            return false;
        }
    }
    return true;
}

int asyncAPImain(int argc, char *argv[]) {
    int devID = 0;
    hipDeviceProp_t devProp;
    printf("[%s] - Starting... \n", argv[0]);

    devID = findCudaDevice(argc, (const char **)argv);
    checkCudaErrors(hipGetDeviceProperties(&devProp, devID));
    printf("CUDA device [%s]\n", devProp.name);

    int n = 16 * 1024 * 1024;
    int nbytes = n * (int)sizeof(int);
    int value = 26;

    int *h_a = nullptr;
    checkCudaErrors(hipHostMalloc((void **)&h_a, nbytes));
    memset(h_a, 0, nbytes);

    int *d_a= nullptr;
    checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
    checkCudaErrors(hipMemset(d_a, 255, nbytes));

    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n / threads.x, 1);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface *timer = nullptr;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;

    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice, 0);
    increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
    hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    // do some thing wait for stream
    unsigned long int counter = 0;
    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
    printf("CPU executed %lu iterations while waiting for GPU to finish\n",
           counter);

    // check the output for correctness
    bool bFinalResults = correct_output(h_a, n, value);

    // release resources
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(h_a));
    checkCudaErrors(hipFree(d_a));

    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}