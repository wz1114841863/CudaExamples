#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__constant__ float constData[256];

__device__ float devData;

__device__ float *devPtr;



__global__ void cudaMemPrint() {
    // display the original value
    printf("Device: the value of the global variable is %f\n", devData);
    // alter the value
    devData += 2.0f;
}

void cudaMemoryMain() {
    float data[256] = {0};
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data)));
    checkCudaErrors(hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data)));

    float value = 3.14f;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));

    float* ptr;
    checkCudaErrors(hipMalloc(&ptr, 256 * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(devPtr), &ptr, sizeof(ptr)));

    float *dp = nullptr;
    checkCudaErrors(hipGetSymbolAddress((void **)&dp,devData));
    checkCudaErrors(hipMemcpy(dp, &value, sizeof(float), hipMemcpyHostToDevice));
    cudaMemPrint<<<1, 1>>>();
    checkCudaErrors(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host:   the value changed by the kernel to %f\n", value);
    size_t sz = 0;
    checkCudaErrors(hipGetSymbolSize(&sz, HIP_SYMBOL(constData)));
    printf("Host:   the sizeof devData: %zu\n", sz);
}