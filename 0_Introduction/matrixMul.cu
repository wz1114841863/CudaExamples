#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// Matrix multiplication (CUDA Kernel) on the device: C = A * B
template <int BLOCK_SIZE>
__global__ void MatrixMulCUDA(float *C, float *A, float *B, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub_matrix

}

void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

// Run a simple test of matrix multiplication using CUDA
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA, const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));

    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));

    // cuds stream
    hipStream_t stream;

    // initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

    if (h_C == nullptr) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host mem to device
    checkCudaErrors(
            hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(
            hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

    // setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16) {
        MatrixMulCUDA<16>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }else {
        MatrixMulCUDA<32>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("done \n");
    checkCudaErrors(hipStreamSynchronize(stream));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, stream))

}