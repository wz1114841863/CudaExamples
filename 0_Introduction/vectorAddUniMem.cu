#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

// cuda kernel function
__global__ void vectorAddAddUniMem(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

// host main function
void vectorAddUniMemMain() {
    // cuda error code
    hipError_t err = hipSuccess;

    //  print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(numElements);
    printf("[Vector addition of %d elements] \n", numElements);

    // allocate the unified memory
    float *A = nullptr;
    err = hipMallocManaged((void **)&A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified memory A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *B = nullptr;
    err = hipMallocManaged((void **)&B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified memory B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *C = nullptr;
    err = hipMallocManaged((void **)&C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified memory C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // initialize the host memory
    for (int i = 0; i < numElements; ++i) {
        A[i] = rand() / (float)RAND_MAX;
        B[i] = rand() / (float)RAND_MAX;
    }

    // execute kernel function
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
           threadsPerBlock);

    vectorAddAddUniMem<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // cuda synchronize
    hipDeviceSynchronize();

    // free memory
    err = hipFree(A);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(B);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(C);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
}