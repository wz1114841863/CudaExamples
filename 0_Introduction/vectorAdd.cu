#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

// cuda kernel function
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

// host main function
void vectorAddMain() {
    // cuda error code
    hipError_t err = hipSuccess;

    //  print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(numElements);
    printf("[Vector addition of %d elements] \n", numElements);

    // allocate the host memory
    auto *h_A = (float *)malloc(size);
    auto *h_B = (float *)malloc(size);
    auto *h_C = (float *)malloc(size);

    //  verify that allocations succeeded
    if (h_A == nullptr || h_B == nullptr || h_C == nullptr) {
        fprintf(stderr, "Failed to allocate host vectors! \n");
        exit(EXIT_FAILURE);
    }

    // initialize the host memory
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // allocate the device memory
    float *d_A = nullptr;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_B = nullptr;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_C = nullptr;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // memeory copy：host -> device
    printf("Copy input data from the host memory to the CUDA device \n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector B from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector C from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // execute kernel function
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
           threadsPerBlock);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // memeory copy：device -> host
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test Passed \n");

    // free memory
    free(h_A);
    free(h_B);
    free(h_C);

    err = hipFree(d_A);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
}